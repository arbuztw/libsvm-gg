#include "hip/hip_runtime.h"
#include "svm.h"
#include <cstdio>
#include <vector>
#include <algorithm>
#include "hipsparse.h"

#define INF HUGE_VAL
#define TAU 1e-12
#define Malloc(type,n) (type *)malloc((n)*sizeof(type))

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

void libsvm2CSR(struct svm_node **xs, const int l, int &nnz, double **valX, int **rowX, int **colX)
{
	std::vector<double> vals;
	std::vector<int> rows, cols;
	vals.reserve(l * 1000);
	rows.reserve(l+1);
	cols.reserve(l * 1000);

	int offset = 0;
	for (int i = 0; i < l; i++) {
		rows.push_back(offset);
		svm_node *x = xs[i];
		for (; x->index != -1; ++x) {
			vals.push_back(x->value);
			cols.push_back(x->index);
			++offset;
		}
	}
	rows.push_back(offset);
	nnz = offset;

	hipMalloc(valX, sizeof(double)*vals.size());
	hipMalloc(rowX, sizeof(int)*rows.size());
	hipMalloc(colX, sizeof(int)*cols.size());
	hipMemcpy(*valX, vals.data(), sizeof(double)*vals.size(), hipMemcpyHostToDevice);
	hipMemcpy(*rowX, rows.data(), sizeof(int)*rows.size(), hipMemcpyHostToDevice);
	hipMemcpy(*colX, cols.data(), sizeof(int)*cols.size(), hipMemcpyHostToDevice);
}

__global__ void rbf_kernel(double gamma, int bsize, int msize,
	double *valB, int *rowB, int *colB,
	double *valM, int *rowM, int *colM,
	double *output)
{
	int idxx = blockIdx.x * blockDim.x + threadIdx.x;
	int idxy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idxx < bsize && idxy < msize) {
		int stx = rowB[idxx], edx = rowB[idxx+1];
		int sty = rowM[idxy], edy = rowM[idxy+1];
		double sum = 0;
		while (stx < edx && sty < edy) {
			if (colB[stx] < colM[sty]) {
				sum += valB[stx] * valB[stx];
				stx++;
			}
			else if (colB[stx] > colM[sty]) {
				sum += valM[sty] * valM[sty];
				sty++;
			}
			else {
				double d = valB[stx] - valM[sty];
				sum += d * d;
				++stx;
				++sty;
			}
		}
		while (stx < edx) {
			sum += valB[stx] * valB[stx];
			stx++;
		}
		while (sty < edy) {
			sum += valM[sty] * valM[sty];
			sty++;
		}
		output[idxy+idxx*msize] = exp(-gamma*sum);
	}
}

__global__ void extend(double *dist, int ncluster, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < ncluster*size) {
		dist[idx+ncluster] = dist[idx%ncluster];
	}
}

__global__ void getIndex(double *dist, int ncluster, int size, int *label)
{
	int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if (idx < size) {
		int st = idx * ncluster;
		double mn = dist[st];
		int mi = 0;
		for (int i = 1; i < ncluster; i++) {
			if (dist[st+i] < mn) {
				mn = dist[st+i];
				mi = i;
			}
		}
		label[idx] = mi;
	}
}

void knkmeans_predict_alllevel(const svm_parameter *param, const svm_problem *prob, struct svm_node** sample,
	int **sub_cidx, int **sub_csize, int msize, double **cluster_avg, int **same_cluster_map,
	const int lvl, const int nchild, int **full_cidx, int **full_csize)
{
	for (int i = 1; i < lvl; i++) {
		for (int j = 0 ; j < (int)pow(nchild,i) ; j++) {
			printf("%d ", sub_csize[i][j]);
		}
		printf("\n");
	}
	const int chunk_size = 10000;

	hipsparseHandle_t handle;
	hipsparseCreate(&handle);
	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	int maxcluster = (int)pow(nchild, lvl-1);
	double **valC = Malloc(double*, lvl);
	int **rowC = Malloc(int*, lvl), **colC = Malloc(int*, lvl);
	double *tmpval = Malloc(double, msize);
	int *tmprow = Malloc(int, maxcluster);
	for (int i = 1; i < lvl; i++) {
		int ncluster = (int)pow(nchild, i);
		hipMalloc(&valC[i], sizeof(double)*msize);
		hipMalloc(&rowC[i], sizeof(int)*(ncluster+1));
		hipMalloc(&colC[i], sizeof(int)*msize);
		int offset = 0;
		for (int j = 0; j < ncluster; j++) {
			tmprow[j] = offset;
			for (int k = 0; k < sub_csize[i][j]; k++)
				tmpval[offset+k] = 1.0 / sub_csize[i][j];
			offset += sub_csize[i][j];
		}
		tmprow[ncluster] = offset;
		hipMemcpy(valC[i], tmpval, sizeof(double)*msize, hipMemcpyHostToDevice);
		hipMemcpy(rowC[i], tmprow, sizeof(int)*(ncluster+1), hipMemcpyHostToDevice);
		hipMemcpy(colC[i], sub_cidx[i], sizeof(int)*msize, hipMemcpyHostToDevice);
	}
	free(tmpval); free(tmprow);

	double *valS;
	int *rowS, *colS, nnzS;
	libsvm2CSR(sample, msize, nnzS, &valS, &rowS, &colS); 

	double *dist;
	hipMalloc(&dist, sizeof(double)*maxcluster*chunk_size);

	int **label = Malloc(int*, lvl);
	for (int i = 0; i < lvl; i++)
		label[i] = Malloc(int, prob->l);

	for (int i = 0; i < prob->l; i++)
		full_cidx[0][i] = i;
	full_csize[0][0] = prob->l;
	
	double *K;
	hipMalloc(&K, sizeof(double)*chunk_size*msize);
			
	int *clabel;
	hipMalloc(&clabel, sizeof(int)*chunk_size);

	for (int i = 0; i < prob->l; i += chunk_size) {
		int sz = std::min(chunk_size, prob->l - i);
		double *valX;
		int *rowX, *colX, nnzX;
		libsvm2CSR(&prob->x[i], sz, nnzX, &valX, &rowX, &colX);
		dim3 gdim(CeilDiv(sz,32), CeilDiv(msize,16)), bdim(32,16);
		rbf_kernel<<<gdim, bdim>>>(param->gamma, sz, msize,
				valX, rowX, colX, valS, rowS, colS, K);
		hipFree(valX);
		hipFree(rowX);
		hipFree(colX);

		for (int l = 1; l < lvl; l++) {
			int ncluster = (int)pow(nchild, l);
			hipMemcpy(dist, cluster_avg[l], sizeof(double)*ncluster, hipMemcpyHostToDevice);
			extend<<<CeilDiv(ncluster*(sz-1), 256), 256>>>(dist, ncluster, sz-1);
			const double dtwo = -2, done = 1;
			hipsparseDcsrmm(handle,
				HIPSPARSE_OPERATION_NON_TRANSPOSE,
				ncluster, sz, msize, msize,
				&dtwo, descr,
				valC[l], rowC[l], colC[l], K, msize, &done, dist, ncluster);
			getIndex<<<CeilDiv(sz, 256), 256>>>(dist, ncluster, sz, clabel);
			hipMemcpy(&label[l][i], clabel, sizeof(int)*sz, hipMemcpyDeviceToHost);
		}
	}
	hipFree(valS); hipFree(rowS); hipFree(colS);
	hipFree(dist); hipFree(K); hipFree(clabel);
	for (int i = 1; i < lvl; i++) {
		hipFree(valC[i]); hipFree(rowC[i]); hipFree(colC[i]);
	}
	free(valC); free(rowC); free(colC);
		
	for (int l = 2; l < lvl; l++) {
		for (int i = 0; i < prob->l; i++) {
			const int cid = same_cluster_map[l-1][label[l-1][i]];
			if (cid > 0) {
				label[l-1][i] = -1;
				label[l][i] = cid;
			}
		}
	}
	
	int *cur_start = Malloc(int, (int)pow(nchild, lvl-1));
	for (int l = 1; l < lvl; l++) {
		int ncluster = (int)pow(nchild, l);
		for (int i = 0; i < ncluster; i++)
			full_csize[l][i] = 0;
		for (int i = 0; i < prob->l; i++) {
			if (label[l][i] < 0) continue;
			full_csize[l][label[l][i]]++;
		}
		int sum = 0;
		for (int i = 0; i < ncluster; i++) {
			cur_start[i] = sum;
			sum += full_csize[l][i];
		}
		for (int i = 0; i < prob->l; i++) {
			if (label[l][i] < 0) continue;
			int &idx = cur_start[label[l][i]];
			full_cidx[l][idx] = i;
			idx++;
		}
	}

	for (int i = 0; i < lvl; i++) free(label[i]);
	free(label);
	free(cur_start);
}

